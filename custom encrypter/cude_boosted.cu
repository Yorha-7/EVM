#include <iostream>
#include <hip/hip_runtime.h>

__device__ bool check_key(const char* candidate, const char* target, int length) {
    for (int i = 0; i < length; i++) {
        if (candidate[i] != target[i]) return false;
    }
    return true;
}

__global__ void brute_force(char* target, int length, bool* found, int* index) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // unique thread id

    // Each thread generates one candidate
    char candidate[5];  // 4 chars + null
    int base = 95;      // ASCII 32–126

    int n = id;
    for (int i = length - 1; i >= 0; i--) {
        candidate[i] = (char)(32 + (n % base));
        n /= base;
    }
    candidate[length] = '\0';

    if (check_key(candidate, target, length)) {
        *found = true;
        *index = id;
    }
}

int main() {
    const char* target = "abcd";
    int length = 4;

    // GPU memory
    char* d_target;
    bool* d_found;
    int* d_index;
    bool h_found = false;
    int h_index = -1;

    hipMalloc(&d_target, length);
    hipMalloc(&d_found, sizeof(bool));
    hipMalloc(&d_index, sizeof(int));

    hipMemcpy(d_target, target, length, hipMemcpyHostToDevice);
    hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (95*95*95*95 + threads - 1) / threads; // enough threads

    brute_force<<<blocks, threads>>>(d_target, length, d_found, d_index);

    hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(&h_index, d_index, sizeof(int), hipMemcpyDeviceToHost);

    if (h_found) {
        std::cout << "Key found at index: " << h_index << std::endl;
    }

    hipFree(d_target);
    hipFree(d_found);
    hipFree(d_index);
    return 0;
}
